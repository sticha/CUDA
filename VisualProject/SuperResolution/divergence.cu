#include "hip/hip_runtime.h"
#include "divergence.h"

__device__ void gradient(float* d_u, float* d_v1, float* d_v2, int x, int y, int c, int w, int h){
	int ind = x + y*w + c*w*h;

	if (x == w - 1) {
		d_v1[ind] = 0.f;
	} else {
		d_v1[ind] = d_u[ind + 1] - d_u[ind];
	}

	if (y == h - 1) {
		d_v2[ind] = 0.f;
	}
	else {
		d_v2[ind] = d_u[ind + w] - d_u[ind];
	}
}

__device__ float2 gradient(float* d_u, int x, int y, int c, int w, int h) {
	int ind = x + y*w + c*w*h;
	float2 ret;

	if (x == w - 1) {
		ret.x = 0.f;
	}
	else {
		ret.x = d_u[ind + 1] - d_u[ind];
	}

	if (y == h - 1) {
		ret.y = 0.f;
	}
	else {
		ret.y = d_u[ind + w] - d_u[ind];
	}

	return ret;
}
