#include "hip/hip_runtime.h"
#include "flow_color.h"
#include <math.h>

__device__ float d_getAngleFromVector(float v1, float v2) {
	// compute angle in radians between motion vector v and (0, 1)
	// the component v2 is assumed to be normalized w.r.t. the original vector v
	float angle = acosf(v2);
	if (v1 < 0) {
		angle = 2 * PI - angle;
	}
	return angle;
}

__global__ void createColorCoding(float* d_v1, float* d_v2, float* d_out, int w, int h, int border, float scale) {
	// get current thread index (x, y)
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	// return if coordinate (x, y) not inside image
	if (x >= w || y >= h) {
		return;
	}

	// index for access image pixel of image with border
	int idxb = x + w * y;
	// index for access image pixel inside image without border
	int idx = (x-border) + (w-2*border) * (y-border);

	// compute vector length
	float v1, v2;
	if (x < border || x >= w - border || y < border || y >= h - border) {
		v1 = (x - w / 2.0f) / (fminf(w, h) * scale / 3.0f);
		v2 = (y - h / 2.0f) / (fminf(w, h) * scale / 3.0f);
	} else {
		v1 = d_v1[idx];
		v2 = d_v2[idx];
	}
	float v_len = sqrtf(v1*v1 + v2*v2);

	if (v_len > EPSILON) {
		// compute angle
		float angle = d_getAngleFromVector(v1, v2 / v_len);

		// use weighted v_len for speed
		v_len *= scale;

		// get color index and color interpolant
		float colorInterp = angle * 3 / PI;
		int colorIdx = static_cast<int>(colorInterp);
		colorInterp -= colorIdx;

		// apply color scheme to output image
		const float intensities[] = { 1.0f, 1.0f, 0.0f, 0.0f, 0.0f, 1.0f };
		float red = intensities[colorIdx] + colorInterp * (intensities[(colorIdx + 1) % 6] - intensities[colorIdx]);
		float green = intensities[(colorIdx + 2) % 6] + colorInterp * (intensities[(colorIdx + 3) % 6] - intensities[(colorIdx + 2) % 6]);
		float blue = intensities[(colorIdx + 4) % 6] + colorInterp * (intensities[(colorIdx + 5) % 6] - intensities[(colorIdx + 4) % 6]);
		d_out[idxb] = fminf(1.0f, v_len*red);
		d_out[idxb + w*h] = fminf(1.0f, v_len*green);
		d_out[idxb + 2 * w*h] = fminf(1.0f, v_len*blue);
	} else {
		// vector to short for beeing color coded
		d_out[idxb] = 0.0f;
		d_out[idxb + w*h] = 0.0f;
		d_out[idxb + 2 * w*h] = 0.0f;
	}
}

__global__ void createColorCoding(float* d_in, float* d_v1, float* d_v2, float* d_out, int w, int h, int nc, int border, float imgVisibility, float scale) {
	// get current thread index (x, y)
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	// return if coordinate (x, y) not inside image
	if (x >= w || y >= h) {
		return;
	}

	bool isBorder = (x < border || x >= w - border || y < border || y >= h - border);

	// width without border
	int wfree = w - 2 * border;
	// height without border
	int hfree = h - 2 * border;

	// index for access image pixel of image with border
	int idxb = x + w * y;
	// index for access image pixel inside image without border
	int idx = (x - border) + wfree * (y - border);

	// compute vector length
	float v1, v2;
	if (isBorder) {
		v1 = (x - w / 2.0f) / (fminf(w, h) * scale / 2.0f);
		v2 = (y - h / 2.0f) / (fminf(w, h) * scale / 2.0f);
	} else {
		v1 = d_v1[idx];
		v2 = d_v2[idx];
	}
	float v_len = sqrtf(v1*v1 + v2*v2);

	// get input image color values
	float in_r, in_g, in_b;
	if (!isBorder) {
		in_r = d_in[idx];
		in_g = in_r;
		in_b = in_r;
		if (nc == 3) {
			in_g = d_in[idx + wfree*hfree];
			in_b = d_in[idx + 2 * wfree*hfree];
		}
	}
	if (v_len > EPSILON) {
		// compute angle
		float angle = d_getAngleFromVector(v1, v2 / v_len);

		// use weighted v_len for speed
		v_len *= scale;

		// get color index and color interpolant
		float colorInterp = angle * 3 / PI;
		int colorIdx = static_cast<int>(colorInterp);
		colorInterp -= colorIdx;

		// apply color scheme to output image (merge with input image data)
		const float intensities[] = { 1.0f, 1.0f, 0.0f, 0.0f, 0.0f, 1.0f };
		float red = intensities[colorIdx] + colorInterp * (intensities[(colorIdx + 1) % 6] - intensities[colorIdx]);
		float green = intensities[(colorIdx + 2) % 6] + colorInterp * (intensities[(colorIdx + 3) % 6] - intensities[(colorIdx + 2) % 6]);
		float blue = intensities[(colorIdx + 4) % 6] + colorInterp * (intensities[(colorIdx + 5) % 6] - intensities[(colorIdx + 4) % 6]);
		if (isBorder) {
			d_out[idxb] = fminf(1.0f, v_len*red);
			d_out[idxb + w*h] = fminf(1.0f, v_len*green);
			d_out[idxb + 2 * w*h] = fminf(1.0f, v_len*blue);
		} else {
			d_out[idxb] = fminf(1.0f, (1 - imgVisibility)*v_len*red + imgVisibility*in_r);
			d_out[idxb + w*h] = fminf(1.0f, (1 - imgVisibility)*v_len*green + imgVisibility*in_g);
			d_out[idxb + 2 * w*h] = fminf(1.0f, (1 - imgVisibility)*v_len*blue + imgVisibility*in_b);
		}
	} else {
		// vector is to short for being color coded
		d_out[idxb] = imgVisibility * in_r;
		d_out[idxb + w*h] = imgVisibility * in_g;
		d_out[idxb + 2 * w*h] = imgVisibility * in_b;
	}
}