#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2015, September 7 - October 6
// ###
// ###
// ### Thomas Moellenhoff, Robert Maier, Caner Hazirbas
// ###
// ###
// ###


#include "helper.h"
#include "divergence.h"
#include "projections.h"
#include "imageTransform.h"
#include "update.h"
#include "updateSuperResolution.h"
#include "flow_color.h"
#include "energy.h"
#include <iostream>
#include <sstream>
#include <iomanip>
using namespace std;


// Default input parameters
const string stdImgPath = "../../material/Images/";
const string stdImgName = "carwide_";
const string stdImgType = "png";
const int stdNumDigits = 2;
const int stdNumImgs = 2;
const int stdStartImg = 1;

// uncomment to use the camera
// #define CAMERA

// uncomment to compute flow field energy
// #define FLOW_ENERGY

// uncomment to compute super resolution energy
// #define SUPER_ENERGY


// struct to transport the pointer for data access on GPU memory
struct Data {

	float**	 d_f;		// [w_small * h_small * nc]: first low resolution input image f1

	float**	 d_u;		// [w * h * nc]: first high resolution output image u1 (also used for intermediate results in optimization process)
	float*   d_Au;		// [w_small * h_small * nc]: Blurred and downsampled version of a high resolution image used for one update step

	float**	 d_v1;		// [w * h]: x-direction of the final flow field v1 (also used for intermediate results in optimization process)
	float**	 d_v2;		// [w * h]: y-direction of the final flow field v2 (also used for intermediate results in optimization process)
	
	float**	 d_b;		// [w * h * nc]: defined as b = u2 - u1 for fix u1, u2 while flow field optimization
	float2** d_A;		// [w * h * nc * 2]: defined as A = gradient(u2) for fix d_u2 while flow field optimization

	float**	 d_v_p;		// [w * h * nc]: dual variable p used for maximization of <p, Av + b> in flow field optimization
	float2** d_v_q1;		// [w * h * 2]: dual variable q1 used for maximization of <q1, gradient(v1)> in flow field optimization
	float2** d_v_q2;		// [w * h * 2]: dual variable q2 used for maximization of <q2, gradient(v2)> in flow field optimization
	
	float**	 d_u_p;		// [w_small * h_small * nc]: dual variable p1 used for maximization of <p1, Au1 - f1> in super resolution optimization
	float**  d_u_Atp;	// [w * h * nc]: Upsampled and Blurred version of p1
	float2** d_u_q;		// [w * h * nc * 2]: dual variable q1 used for maximization of <q1, gradient(u1)> in super resolution optimization

	float**	 d_u_r;		// [w * h * nc]: dual variable r used for maximization of <r, Bu> in super resolution optimization

	float*   d_temp_big;	// [w * h * nc]: Intermediate result of a big sized image

	float**  d_flow;		// [(w + 2 * border) * (h + 2 * border) * 3]: stores the color coded final flow field as an output image


#if defined(FLOW_ENERGY) || defined(SUPER_ENERGY)
	float* d_energy;	// stores in a single value the energy of the previous calculated flow field
#endif

};


// Functions for GPU calculations

// Allocate memory on GPU for all arrays that are used for calculation
void allocateGPUMemory(Data& data, int numImgs, int w, int h, int w_small, int h_small, int nc, int colorBorder) {
	// Helper values
	size_t n_small = w_small*h_small*nc;
	size_t n = w*h*nc;
	int wborder = w + 2 * colorBorder;
	int hborder = h + 2 * colorBorder;

	hipMalloc(&data.d_temp_big, n*sizeof(float));
	CUDA_CHECK;
	hipMalloc(&data.d_Au, n_small*sizeof(float));
	CUDA_CHECK;

#if defined(FLOW_ENERGY) || defined(SUPER_ENERGY)
	hipMalloc(&data.d_energy, sizeof(float));
	CUDA_CHECK;
#endif

	data.d_f = new float*[numImgs];
	data.d_u = new float*[numImgs];
	data.d_v1 = new float*[numImgs - 1];
	data.d_v2 = new float*[numImgs - 1];
	data.d_b = new float*[numImgs - 1];
	data.d_A = new float2*[numImgs-1];
	data.d_v_p = new float*[numImgs - 1];
	data.d_v_q1 = new float2*[numImgs - 1];
	data.d_v_q2 = new float2*[numImgs - 1];
	data.d_u_p = new float*[numImgs];
	data.d_u_Atp = new float*[numImgs];
	data.d_u_q = new float2*[numImgs];
	data.d_u_r = new float*[numImgs - 1];
	data.d_flow = new float*[numImgs];


	for (int i = 0; i < numImgs; i++) {
		// # Allocate GPU memory
		hipMalloc(&data.d_f[i], n_small*sizeof(float));
		CUDA_CHECK;
		hipMalloc(&data.d_u[i], n*sizeof(float));
		CUDA_CHECK;
		hipMalloc(&data.d_u_p[i], n_small*sizeof(float));
		CUDA_CHECK;
		hipMalloc(&data.d_u_Atp[i], n*sizeof(float));
		CUDA_CHECK;	
		hipMalloc(&data.d_u_q[i], n*sizeof(float2));
		CUDA_CHECK;
	}

	for (int i = 0; i < numImgs - 1; i++) {
		hipMalloc(&data.d_v1[i], w*h*sizeof(float));
		CUDA_CHECK;
		hipMalloc(&data.d_v2[i], w*h*sizeof(float));
		CUDA_CHECK;
		hipMalloc(&data.d_b[i], n*sizeof(float));
		CUDA_CHECK;
		hipMalloc(&data.d_v_p[i], n*sizeof(float));
		CUDA_CHECK;
		hipMalloc(&data.d_A[i], n*sizeof(float2));
		CUDA_CHECK;
		hipMalloc(&data.d_v_q1[i], w*h*sizeof(float2));
		CUDA_CHECK;
		hipMalloc(&data.d_v_q2[i], w*h*sizeof(float2));
		CUDA_CHECK;
		hipMalloc(&data.d_flow[i], wborder*hborder * 3 * sizeof(float));
		CUDA_CHECK;
		hipMalloc(&data.d_u_r[i], n*sizeof(float));
		CUDA_CHECK;
	}
	
}

// Initializes the arrays on GPU memory for optimization process
void InitializeGPUData(float** f, Data& data, int numImgs, int w, int h, int w_small, int h_small, int nc) {
	// Helper values
	size_t n_small = w_small*h_small*nc;
	size_t n = w * h * nc;
	
	// Calculate grid size
	dim3 block3d = dim3(16, 16, nc);
	dim3 grid3d = dim3((w + block3d.x - 1) / block3d.x, (h + block3d.y - 1) / block3d.y, 1);

	int smBytes = (block3d.x + 4) * (block3d.y + 4) * sizeof(float);
	for (int i = 0; i < numImgs; i++) {
		hipMemset(data.d_u_p[i], 0, n_small*sizeof(float));
		CUDA_CHECK;
		// Copy images to GPU
		hipMemcpy(data.d_f[i], f[i], n_small * sizeof(float), hipMemcpyHostToDevice);
		CUDA_CHECK;
		// Upsample f to v_p (temporary result) and blur v_p to u
		initialUpsample<<<grid3d, block3d>>>(data.d_f[i], data.d_temp_big, w, h, w_small, h_small);
		hipDeviceSynchronize();
		CUDA_CHECK;
		gaussBlur5<<<grid3d, block3d, smBytes>>>(data.d_temp_big, data.d_u[i], w, h);
		hipDeviceSynchronize();
		CUDA_CHECK;

		hipMemset(data.d_u_q[i], 0, n*sizeof(float2));
		CUDA_CHECK;
	}

	for (int i = 0; i < numImgs - 1; i++) {
		// Fill arrays with 0
		hipMemset(data.d_v1[i], 0, w*h*sizeof(float));
		CUDA_CHECK;
		hipMemset(data.d_v2[i], 0, w*h*sizeof(float));
		CUDA_CHECK;
		// Initialize dual variables
		hipMemset(data.d_v_p[i], 0, w*h*nc*sizeof(float));
		CUDA_CHECK;
		hipMemset(data.d_v_q1[i], 0, w*h*sizeof(float2));
		CUDA_CHECK;
		hipMemset(data.d_v_q2[i], 0, w*h*sizeof(float2));
		CUDA_CHECK;
		hipMemset(data.d_u_r[i], 0, n*sizeof(float));
		CUDA_CHECK;
	}	
}

// Free all allocated GPU memory
void freeGPUMemory(Data& data, int numImgs) {
	hipFree(data.d_Au);
	hipFree(data.d_temp_big);
	for (int i = 0; i < numImgs; i++) {
		hipFree(data.d_f[i]);
		hipFree(data.d_u[i]);
		hipFree(data.d_u_p[i]);
		hipFree(data.d_u_Atp[i]);
		hipFree(data.d_u_q[i]);
	}
	for (int i = 0; i < numImgs - 1; i++) {
		hipFree(data.d_b[i]);
		hipFree(data.d_v1[i]);
		hipFree(data.d_v2[i]);
		hipFree(data.d_v_p[i]);
		hipFree(data.d_A[i]);
		hipFree(data.d_v_q1[i]);
		hipFree(data.d_v_q2[i]);
		hipFree(data.d_flow[i]);
		hipFree(data.d_u_r[i]);
	}
	
#if defined(FLOW_ENERGY) || defined(SUPER_ENERGY)
	hipFree(data.d_energy);
#endif
	
	CUDA_CHECK;
}

// Computes the flow field (v1, v2) for fixed images u1 and u2
void calculateFlow(Data& data, int numImgs, float gamma, int iterations, int w, int h, int nc) {
	// Calculate grid size
	dim3 block3d = dim3(16, 16, nc);
	dim3 grid3d = dim3((w + block3d.x - 1) / block3d.x, (h + block3d.y - 1) / block3d.y, 1);

	dim3 block2d = dim3(16, 16, 1);
	dim3 grid2d = dim3((w + block2d.x - 1) / block2d.x, (h + block2d.y - 1) / block2d.y, 1);

#ifdef FLOW_ENERGY
	dim3 block1d = dim3(128, 1, 1);
	dim3 grid1d = dim3((w*h + block1d.x - 1) / block1d.x, 1, 1);
	int bytesSM1d = block1d.x * sizeof(float);
#endif

	for (int i = 0; i < numImgs - 1; i++) {
		// Compute b = u2 - u1
		imageDiff<<<grid3d, block3d>>>(data.d_u[i], data.d_u[i + 1], data.d_b[i], w, h);
		hipDeviceSynchronize();
		CUDA_CHECK;
		// Compute A = gradient(u2)
		calculateGradientCD<<<grid3d, block3d>>>(data.d_u[i + 1], data.d_A[i], w, h, nc);
		hipDeviceSynchronize();
		CUDA_CHECK;
	}

	// Step sizes
	float sigmaQ = 0.5f;

	// Update in an alternating fashion the dual variables p, q1, q2 and the primal variable (flow field) v
	for (int i = 0; i < iterations; i++) {
		
		for (int i = 0; i < numImgs - 1; i++) {
			// Update dual variable p
			flow_updateP<<<grid3d, block3d>>>(data.d_v_p[i], data.d_v1[i], data.d_v2[i], data.d_A[i], data.d_b[i], gamma, w, h);
			hipDeviceSynchronize();
			CUDA_CHECK;
			// Update dual variable q1
			flow_updateQ<<<grid2d, block2d>>>(data.d_v_q1[i], data.d_v1[i], sigmaQ, w, h);
			hipDeviceSynchronize();
			CUDA_CHECK;
			// Update dual variable q2
			flow_updateQ<<<grid2d, block2d>> >(data.d_v_q2[i], data.d_v2[i], sigmaQ, w, h);
			hipDeviceSynchronize();
			CUDA_CHECK;
			// Update flow field v
			flow_updateV<<<grid2d, block2d>>>(data.d_v1[i], data.d_v2[i], data.d_v_p[i], data.d_v_q1[i], data.d_v_q2[i], data.d_A[i], w, h, nc);
			hipDeviceSynchronize();
			CUDA_CHECK; 
		}

#ifdef FLOW_ENERGY
		// Compute energy of latest flow field
		hipMemset(data.d_energy, 0, sizeof(float));
		CUDA_CHECK;
		flowFieldEnergy<<<grid1d, block1d, bytesSM1d>>>(data.d_energy, data.d_A, data.d_b, data.d_v1, data.d_v2, gamma, w, h, nc);
		hipDeviceSynchronize();
		CUDA_CHECK;
		float energy;
		hipMemcpy(&energy, data.d_energy, sizeof(float), hipMemcpyDeviceToHost);
		CUDA_CHECK;
		cout << "Flow field energy in iteration " << i << ": " << energy << endl;
#endif
	}
}

// Computes super resolution images u1, u2 for fixed flow field (v1, v2)
void calculateSuperResolution(Data& data, int numImgs, int iterations, float alpha, float beta, float gamma, int w, int h, int w_small, int h_small, int nc) {
	// Helper values
	int n = w*h*nc;
	int n_small = w_small*h_small*nc;

	// Calculate grid size
	dim3 block3d = dim3(16, 16, nc);
	dim3 grid3d = dim3((w + block3d.x - 1) / block3d.x, (h + block3d.y - 1) / block3d.y, 1);
	dim3 grid3d_small = dim3((w_small + block3d.x - 1) / block3d.x, (h_small + block3d.y - 1) / block3d.y, 1);

	dim3 block2d = dim3(16, 16, 1);
	dim3 grid2d = dim3((w + block2d.x - 1) / block2d.x, (h + block2d.y - 1) / block2d.y, 1);

	int smBytes = (block3d.x + 4) * (block3d.y + 4) * sizeof(float);

#ifdef SUPER_ENERGY
	dim3 block1d = dim3(128, 1, 1);
	dim3 grid1d = dim3((w*h + block1d.x - 1) / block1d.x, 1, 1);
	int bytesSM1d = block1d.x * sizeof(float);
#endif

	// Step sizes
	float sigmaP = 1.0f;
	float sigmaQ = 0.5f;

	// Update in an alternating fashion the dual variables p1, p2, q1, q2, r and the primal variables (super resolution images) u1, u2
	for (int i = 0; i < iterations; i++) {
		for (int i = 0; i < numImgs; i++) {
			// Blur u1
			gaussBlur5<<<grid3d, block3d, smBytes>>>(data.d_u[i], data.d_temp_big, w, h);
			hipDeviceSynchronize();
			CUDA_CHECK;
			// Downsample blurred u1
			downsample<<<grid3d_small, block3d>>>(data.d_temp_big, data.d_Au, w, h, w_small, h_small);
			hipDeviceSynchronize();
			CUDA_CHECK;
			// Update dual variable p1
			super_updateP<<<grid3d_small, block3d>>>(data.d_u_p[i], data.d_f[i], data.d_Au, sigmaP, alpha, w_small, h_small);
			hipDeviceSynchronize();
			CUDA_CHECK;
			// Update dual variable q1
			super_updateQ<<<grid3d, block3d>>>(data.d_u_q[i], data.d_u[i], sigmaQ, beta, w, h);
			hipDeviceSynchronize();
			CUDA_CHECK;
			// Upsample p1
			upsample<<<grid3d, block3d>>>(data.d_u_p[i], data.d_temp_big, w, h, w_small, h_small);
			hipDeviceSynchronize();
			CUDA_CHECK;
			// Blur upsampled p1
			gaussBlur5<<<grid3d, block3d, smBytes>>>(data.d_temp_big, data.d_u_Atp[i], w, h);
			hipDeviceSynchronize();
			CUDA_CHECK;
		}
		for(int i = 0; i < numImgs - 1;i++){
			// Update dual variable r
			super_updateR<<<grid3d, block3d>>>(data.d_u_r[i], data.d_u[i], data.d_u[i+1], data.d_v1[i], data.d_v2[i], gamma, w, h);
			hipDeviceSynchronize();
			CUDA_CHECK;
			// Update super resolution images u1, u2
			super_updateU<<<grid3d, block3d>>>(data.d_u[i], data.d_u[i+1], data.d_u_r[i], data.d_u_Atp[i], data.d_u_Atp[i+1], data.d_u_q[i], data.d_u_q[i+1], data.d_v1[i], data.d_v2[i], w, h);
			hipDeviceSynchronize();
			CUDA_CHECK;
		}
		
		
#ifdef SUPER_ENERGY
		// Compute energy of latest super resolution
		hipMemset(data.d_energy, 0, sizeof(float));
		CUDA_CHECK;
		superResolutionEnergy<<<grid1d, block1d, bytesSM1d>>>(data.d_energy, data.d_u1, data.d_u2, data.d_f1, data.d_f2, data.d_v1, data.d_v2, alpha, beta, gamma, w, h, nc);
		hipDeviceSynchronize();
		CUDA_CHECK;
		float energy;
		hipMemcpy(&energy, data.d_energy, sizeof(float), hipMemcpyDeviceToHost);
		CUDA_CHECK;
		cout << "Super resolution energy in iteration " << i << ": " << energy << endl;
#endif
	}
}

// Get the results from the calculation
void getComputationResult(Data& data, int numImgs, float** v1, float** v2, float** flow, float** sr, int w, int h, int nc, int colorBorder) {
	// Helper values
	int wborder = w + 2 * colorBorder;
	int hborder = h + 2 * colorBorder;

	// Calculate grid size
	dim3 block2d = dim3(16, 16, 1);
	dim3 grid2d = dim3((w + block2d.x - 1) / block2d.x, (h + block2d.y - 1) / block2d.y, 1);

	dim3 grid2dborder = dim3((wborder + block2d.x - 1) / block2d.x, (hborder + block2d.y - 1) / block2d.y, 1);

	for (int i = 0; i < numImgs - 1; i++) {
		// Generate a color coding for the flow field
		createColorCoding<<<grid2dborder, block2d>>>(data.d_v1[i], data.d_v2[i], data.d_flow[i], wborder, hborder, colorBorder);
		//createColorCoding<<<grid2dborder, block2d>>>(data.d_u1, data.d_v1, data.d_v2, data.d_flow, wborder, hborder, nc, colorBorder);
		hipDeviceSynchronize();
		CUDA_CHECK;

		// Copy results to Host
		hipMemcpy(v1[i], data.d_v1[i], w * h * sizeof(float), hipMemcpyDeviceToHost);
		CUDA_CHECK;
		hipMemcpy(v2[i], data.d_v2[i], w * h * sizeof(float), hipMemcpyDeviceToHost);
		CUDA_CHECK;
		hipMemcpy(flow[i], data.d_flow[i], wborder * hborder * 3 * sizeof(float), hipMemcpyDeviceToHost);
		CUDA_CHECK;
	}
	for (int i = 0; i < numImgs; i++) {
		hipMemcpy(sr[i], data.d_u[i], w * h * nc * sizeof(float), hipMemcpyDeviceToHost);
		CUDA_CHECK;
	}
}


int main(int argc, char **argv) {
	// Before the GPU can process your kernels, a so called "CUDA context" must be initialized
	// This happens on the very first call to a CUDA function, and takes some time (around half a second)
	// We will do it right here, so that the run time measurements are accurate
	hipDeviceSynchronize();  CUDA_CHECK;


	// Reading command line parameters:
	// getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
	// If "-param" is not specified, the value of "var" remains unchanged
	//
	// return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise


	// Number of computation repetitions to get a better run time measurement
	int repeats = 1;
	getParam("repeats", repeats, argc, argv);
	cout << "repeats: " << repeats << endl;

	// Load the input image as grayscale if "-gray" is specifed
	bool gray = false;
	getParam("gray", gray, argc, argv);
	cout << "gray: " << gray << endl;

	// Value for tuning similarity of the downsampled high resolution results to the low resolution input images
	float alpha = 1.f;
	getParam("alpha", alpha, argc, argv);
	cout << "alpha: " << alpha << endl;

	// Value for tuning the total variation of the high resolution images
	float beta = 0.01f;
	getParam("beta", beta, argc, argv);
	cout << "beta: " << beta << endl;

	// Value for tuning the importance of the flow constraint
	float gamma = 8.f;
	getParam("gamma", gamma, argc, argv);
	cout << "gamma: " << gamma << endl;

	// Thickness of the colored border in the output image of the color coded flow field
	int colorBorder = 4;
	getParam("border", colorBorder, argc, argv);
	cout << "color coding border: " << colorBorder << endl;

	// Number of iterations for each update of the flow field and the super resolution images
	int iterations = 200;
	getParam("iterations", iterations, argc, argv);
	cout << "iterations: " << iterations << endl;

	// Path to the low resolution input images
	string imgPath = stdImgPath;
	getParam("path", imgPath, argc, argv);
	cout << "Path to the images: " << imgPath << endl;

	// Common part of the name of the low resolution input images
	string imgName = stdImgName;
	getParam("name", imgName, argc, argv);
	cout << "Name of the images: " << imgName << endl;

	// Type of the input images (e.g. "png")
	string imgType = stdImgType;
	getParam("type", imgType, argc, argv);
	cout << "Type of the images: " << imgType << endl;
	imgType = "." + imgType;

	// Number of input images to load
	int numImgs = stdNumImgs;
	getParam("count", numImgs, argc, argv);
	cout << "How many images to load: " << numImgs << endl;

	// Number of digits that should be respected in loading the input images
	int numDigits = stdNumDigits;
	getParam("digits", numDigits, argc, argv);
	cout << "Number of digits in the name: " << numDigits << endl;

	// Index of the first input image to load
	int startImg = stdStartImg;
	getParam("start", startImg, argc, argv);
	cout << "Index of the start Image: " << startImg << endl;

	// Init camera / Load input image

#ifdef CAMERA
	// Init camera
	cv::VideoCapture camera(0);
	if (!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
	int camW = 640;
	int camH = 480;
	camera.set(CV_CAP_PROP_FRAME_WIDTH, camW);
	camera.set(CV_CAP_PROP_FRAME_HEIGHT, camH);
	// Read in first frame to get the dimensions
	numImgs = 2;
	cv::Mat* mIn = new cv::Mat[numImgs];
	camera >> mIn[0];
	camera >> mIn[1];
	// Convert to float representation (opencv loads image values as single bytes by default)
	mIn[0].convertTo(mIn[0], CV_32F);
	mIn[1].convertTo(mIn[1], CV_32F);
	// Convert range of each channel from [0, 255] to [0, 1]
	mIn[0] /= 255.f;
	mIn[1] /= 255.f;

#else
	// Load all of the images needed
	cv::Mat* mIn = new cv::Mat[numImgs];
	for (int i = 0; i < numImgs; i++){
		int imageIdx = startImg + i;
		// Generating the complete image path
		stringstream ss;
		ss << setw(numDigits) << setfill('0') << imageIdx;
		string image = imgPath + imgName + ss.str() + imgType;
		// Loading the image
		mIn[i] = cv::imread(image.c_str(), (gray ? CV_LOAD_IMAGE_GRAYSCALE : -1));
		if (mIn[i].data == NULL) {
			cerr << "ERROR: Could not load image " << image << endl;
			system("pause");
			return 1;
		}
		// Convert to float representation (opencv loads image values as single bytes by default)
		mIn[i].convertTo(mIn[i], CV_32F);
		// Convert range of each channel from [0, 255] to [0, 1]
		mIn[i] /= 255.f;
	}
#endif

	// Get the dimensions of the images
	int w_small = mIn[0].cols;
	int h_small = mIn[0].rows;
	int w = 2 * w_small;
	int h = 2 * h_small;
	int nc = mIn[0].channels();
	cout << "input images: " << w_small << " x " << h_small << endl;

	
	// Set the output image format
	cv::Mat mSR(h, w, mIn[0].type());
	cv::Mat mFlow((h + 2 * colorBorder), (w + 2 * colorBorder), CV_32FC3);	
	cv::Mat mV1(h, w, CV_32FC1);
	cv::Mat mV2(h, w, CV_32FC1);

	//cv::Mat mOut(h, w, mIn.type());  // mOut will have the same number of channels as the input image, nc layers
	//cv::Mat mOut(h, w, CV_32FC3);    // mOut will be a color image, 3 layers
	//cv::Mat mOut(h, w, CV_32FC1);    // mOut will be a grayscale image, 1 layer

	// Allocate memory for an arbitary amount of input images
	float** imgIn = new float*[numImgs];
	float** imgFlow = new float*[numImgs - 1];
	float** imgV1 = new float*[numImgs - 1];
	float** imgV2 = new float*[numImgs - 1];
	float** imgSR = new float*[numImgs];

	for (int i = 0; i < numImgs; i++){
		imgIn[i] = new float[(size_t)(w_small*h_small*nc)];
		imgSR[i] = new float[(size_t)w*h*mSR.channels()];
	}

	for (int i = 0; i < numImgs - 1; i++) {
		// Allocate memory for output images
		imgFlow[i] = new float[(size_t)(w + 2 * colorBorder)*(h + 2 * colorBorder)*mFlow.channels()];
		imgV1[i] = new float[(size_t)w*h*mV1.channels()];
		imgV2[i] = new float[(size_t)w*h*mV2.channels()];
	}
	


#ifdef CAMERA

	Data data;
	// Allocate memory for gpu arrays
	allocateGPUMemory(data, w, h, 2, w_small, h_small, nc, colorBorder);

	// Read a camera image frame every 30 milliseconds:
	// cv::waitKey(30) waits 30 milliseconds for a keyboard input,
	// returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
	while (cv::waitKey(30) < 0) {
		float* temp = imgIn[0];
		imgIn[0] = imgIn[1];
		imgIn[1] = temp;
		// Get camera image
		camera >> mIn[1];
		// convert to float representation (opencv loads image values as single bytes by default)
		mIn[1].convertTo(mIn[1], CV_32F);
		// Convert range of each channel from [0, 255] to [0, 1]
		mIn[1] /= 255.f;

		// Init raw input image array
		// opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
		// But for CUDA it's better to work with layered images: rrr... ggg... bbb...
		// So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
		convert_mat_to_layered(imgIn[1], mIn[1]);
#else
		// Convert all images
		for (int i = 0; i < numImgs; i++) {
			convert_mat_to_layered(imgIn[i], mIn[i]);
		}

		// Allocate memory for gpu arrays
		Data data;
		allocateGPUMemory(data, numImgs, w, h, w_small, h_small, nc, colorBorder);
#endif

		// Fetch time before calculation
		Timer timer;
		timer.start();

		// # Call the CUDA computation
		// Initialize arrays with start values
		InitializeGPUData(imgIn, data, numImgs, w, h, w_small, h_small, nc);
		// Alternating optimization of flow field and super resolution images
		for (int i = 0; i < 1; i++) {
			// Compute flow estimation
			calculateFlow(data, numImgs, gamma, iterations, w, h, nc);
			// Compute super resolution
			calculateSuperResolution(data, numImgs, iterations, alpha, beta, gamma, w, h, w_small, h_small, nc);
		}
		// Get results from computation
		getComputationResult(data, numImgs, imgV1, imgV2, imgFlow, imgSR, w, h, nc, colorBorder);
		
		// Get time after calculation and compute duration
		timer.end();
		float t = timer.get();  // elapsed time in seconds
		cout << "time: " << t * 1000 << " ms" << endl;
		
		// show input image
#ifdef CAMERA
		convert_layered_to_mat(mIn[0], imgIn[0]);
#endif
		// Show input images
		for (int i = 0; i < numImgs; i++) {
			string name = "Input " + to_string(i);
			showImage(name, mIn[i], 100, 100);
			name = "Super Resolution " + to_string(i);
			convert_layered_to_mat(mSR, imgSR[i]);
			showImage(name, mSR, 100 + w_small + 40, 100);
		}
		
		// Show all output images
		for (int i = 0; i < numImgs - 1; i++) {
			convert_layered_to_mat(mV1, imgV1[i]);
			showImage("V1", (mV1 + 1.0f) / 2.0f, 100 + w_small + w + 80, 100);
			convert_layered_to_mat(mV2, imgV2[i]);
			showImage("V2", (mV2 + 1.0f) / 2.0f, 100 + w_small + w + 80, 100);
			convert_layered_to_mat(mFlow, imgFlow[i]);
			showImage("Flow Field", mFlow, 100 + w_small + w + 80, 100);
		}
		

#ifdef CAMERA
		// end of camera loop
	}
	// Free arrays on gpu memory
	freeGPUMemory(data);
#else
	// Free arrays on gpu memory
	freeGPUMemory(data, numImgs);
	// wait for key inputs
	cv::waitKey(0);
#endif


#ifdef SAVE
	// save input and result
	cv::imwrite("image_input.png", mIn*255.f);  // "imwrite" assumes channel range [0,255]
	cv::imwrite("image_V1.png", (mV1 + 1.0f) / 2.0f * 255.f);
	cv::imwrite("image_V2.png", (mV2 + 1.0f) / 2.0f * 255.f);
#endif

	// free allocated arrays
	delete[] imgIn;
	delete[] imgFlow;
	
	// close all opencv windows
	cvDestroyAllWindows();
	return 0;
}



