#include "hip/hip_runtime.h"
#include "update.h"

__global__ void updateP(float* d_p, float* d_v1, float* d_v2, float2* d_A, float* d_b, float sigma, float gamma, int w, int h) {
	// get current thread index (x, y, c)
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockDim.y;
	int c = threadIdx.z;

	// return if coordinate (x, y) not inside image
	if (x >= w || y >= h) {
		return;
	}

	// index for access without color channel
	int idx = x + w * y;
	// index for access with color channel
	int idxc = idx + w * h * c;

	// compute:
	// (1) p_k+1' = proj_C(p_k + s * (Av + b))
	// (2) p_k+1 = 2 * p_k+1' - p_k

	// b
	float acc = d_b[idxc];
	// Av + b
	acc += d_A[idxc].x * d_v1[idx] + d_A[idxc].y * d_v2[idx];
	// p + s * (Av + b)
	float oldp = d_p[idxc];
	acc = oldp + sigma * acc;
	// proj_C(p_k + s * (Av + b))
	acc = projC(acc, gamma);
	// sor: p_k+1 = 2 * p_k+1' - p_k
	d_p[idxc] = 2 * acc - oldp;
}

__global__ void updateQ(float2* d_q, float* d_v, float sigma, int w, int h) {
	// get current thread index (x, y)
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockDim.y;

	// return if coordinate (x, y) not inside image
	if (x >= w || y >= h) {
		return;
	}

	// index for access image pixel
	int idx = x + w * y;

	// compute:
	// (1) q_k+1' = proj_D(q_k + s * (dv/dx dv/dy))
	// (2) q_k+1 = 2 * q_k+1' - q_k

	// dv/dx dv/dy
	float2 acc = gradient(d_v, x, y, 0, w, h);
	// q_k + s * (dv/dx dv/dy)
	float2 qold = d_q[idx];
	acc = make_float2(qold.x + sigma * acc.x, qold.y + sigma * acc.y);
	// proj_D(q_k + s * (dv/dx dv/dy))
	acc = projD(acc);
	// sor: q_k+1 = 2 * q_k+1' - q_k
	d_q[idx] = make_float2(2 * acc.x - qold.x, 2 * acc.y - qold.y);
}

__global__ void updateV(float* d_v1, float* d_v2, float* d_p, float2* d_q1, float2* d_q2, float2* d_A, float tau, int w, int h) {
	// get current thread index (x, y)
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockDim.y;

	// return if coordinate (x, y) not inside image
	if (x >= w || y >= h) {
		return;
	}

	// index for access image pixel
	int idx = x + w * y;

	// compute:
	// (1) v_k+1 = v_k - t * (A * p_k+1 - (div(q1_k+1) div(q2_k+1)))

	// div(q1_k+1)
	float div_q1 = divergence(d_q1, x, y, w, h);
	// div(q2_k+1)
	float div_q2 = divergence(d_q2, x, y, w, h);
	// p_k+1
	float p1 = d_p[idx];
	float p2 = d_p[idx + w*h];
	float p3 = d_p[idx + 2 * w*h];
	// A * p_k+1 - (div(q1_k+1) div(q2_k+1))
	float acc1 = d_A[idx].x * p1 + d_A[idx + w*h].x * p2 + d_A[idx + 2 * w*h].x * p3 - div_q1;
	float acc2 = d_A[idx].y * p1 + d_A[idx + w*h].y * p2 + d_A[idx + 2 * w*h].y * p3 - div_q2;
	// v_k - t * (A * p_k+1 - (div(q1_k+1) div(q2_k+1)))
	d_v1[idx] -= tau * acc1;
	d_v2[idx] -= tau * acc2;
}